#include "hip/hip_runtime.h"

#include <chrono>
#include <cstdint>
#include <iostream>
#include <hip/hip_runtime.h>

#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 600
#else
    __device__ static double atomicAdd(double *addr, double val){
        double old=*addr, assumed;
        do {
            assumed = old;
            old = __longlong_as_double(
            atomicCAS((unsigned long long int*)addr,
              __double_as_longlong(assumed),
              __double_as_longlong(val+assumed) )
            );
        } while (assumed!=old);
        return old;
    }
#endif


static inline void CHECK_CUDA(hipError_t code){
    if (code != hipSuccess){
        std::cout << "A CUDA error check failed." << std::endl;
    }
}


__global__ void reduce_kernel(
    const int64_t num_elements,
    const int64_t num_sources,
    const int64_t num_components,
    const int64_t * RESTRICT d_source_indices,
    const double * RESTRICT d_source_values,
    double * RESTRICT d_elements
){
    const int idx = threadIdx.x + blockIdx.x * blockDim.x; 
    if(idx < num_sources){
        
        const int64_t index = d_source_indices[idx] - 1;
        for (int64_t cx=0 ; cx<num_components ; cx++){
            const int64_t output_index = cx * num_elements + index;
            const double value = d_source_values[cx * num_sources + idx];
            atomicAdd(&d_elements[output_index], value);
        }

    }
    return;
}


extern "C" int c_runner(
    const int64_t num_threads,
    const int64_t num_elements,
    const int64_t num_sources,
    const int64_t num_components,
    const int64_t * RESTRICT source_indices,
    const double * RESTRICT source_values,
    double * RESTRICT elements,
    double * RESTRICT t_internal
){
    

    int64_t *d_source_indices;
    double *d_source_values;
    double *d_elements;

    CHECK_CUDA(hipMalloc(&d_source_indices, num_sources * sizeof(int64_t)));
    CHECK_CUDA(hipMalloc(&d_source_values, num_sources * num_components * sizeof(double)));
    CHECK_CUDA(hipMalloc(&d_elements, num_elements * num_components * sizeof(double)));

    CHECK_CUDA(hipMemcpy(
        d_source_indices, source_indices, num_sources * sizeof(int64_t), hipMemcpyHostToDevice
    ));
    CHECK_CUDA(hipMemcpy(
        d_source_values, source_values, num_sources * num_components * sizeof(double), hipMemcpyHostToDevice
    ));
    CHECK_CUDA(hipMemcpy(
        d_elements, elements, num_elements * num_components * sizeof(double), hipMemcpyHostToDevice
    ));

    std::chrono::high_resolution_clock::time_point _loop_timer_t0 = std::chrono::high_resolution_clock::now();

    
    const int grid_size = 1 + (num_sources / num_threads);
    reduce_kernel<<<grid_size, num_threads>>>(
        num_elements,
        num_sources,
        num_components,
        d_source_indices,
        d_source_values,
        d_elements
    );

    CHECK_CUDA(hipDeviceSynchronize());

    std::chrono::high_resolution_clock::time_point _loop_timer_t1 = std::chrono::high_resolution_clock::now();
 std::chrono::duration<double> _loop_timer_res = _loop_timer_t1 - _loop_timer_t0;
    *t_internal= (double) _loop_timer_res.count();


    CHECK_CUDA(hipMemcpy(
        elements, d_elements, num_elements * num_components * sizeof(double), hipMemcpyDeviceToHost
    ));

    CHECK_CUDA(hipFree(d_source_indices));
    CHECK_CUDA(hipFree(d_source_values));
    CHECK_CUDA(hipFree(d_elements));

    return 0;
}



